#include "hip/hip_runtime.h"
// Created by A. Aichert on Tue Oct 4th 2016
#include <iostream>

#include <LibUtilsCuda/UtilsCuda.hxx>

#include <LibUtilsCuda/CudaMemory.h>
#include <LibUtilsCuda/CudaBindlessTexture.h>

#include <LibUtilsCuda/culaut/culaut.hxx>

template <int n_c>
__global__ void kernel_raycast(
	unsigned short n_u, unsigned short n_v,
	float * pixel_data,				//< image data (RGBA)
	float * model_C_Pinv_h,			//< volume model transform (voxels to world)
	hipTextureObject_t voxel_data,	//< volume data
	float* ray_entry,				//< ray entry
	float* ray_exit,				//< ray exit
	float * noise,					//< noise for ray offsets 
	float   samples_per_voxel		//< samples per voxel
	)
{
	// Find index of current thread
	int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_x>=n_u) return;
	if (idx_y>=n_v) return;
	int idx=idx_y*n_u+idx_x;

	// Access correct pixel
	float *entry=ray_entry+4*idx;
	float *exit =ray_exit+4*idx;
	float *pixel=pixel_data+n_c*idx;

	float ray_length=culaut::xvdistance2<float,3>(entry,exit);
	if (ray_length<=0)
	{
		culaut::xvset<float,n_c>(pixel,0);
		return;
	}

	float intensity=0;
	float step=1.0/ray_length;
	step/=samples_per_voxel;
	float voxel[3];
	for (float a=-noise[idx]*step;a<1.0;a+=step)
	{
		culaut::xvlincomb<float,float,3>(voxel,entry,1.0-a,exit,a);
		float sample=tex3D<float>(voxel_data,voxel[0],voxel[1],voxel[2]);
		if (intensity<sample) intensity=sample;
	}

	culaut::xvset<float,n_c>(pixel,intensity);

}


void raycast_mip(
	int n_u, int n_v, int n_c,		//< image size and number of channels
	float * pixel_data_d,			//< image data (RGBA)
	float * model_C_Pinv_d,			//< volume model transform (voxels to world)
	hipTextureObject_t voxel_data,	//< volume data
	float * ray_entry_d,			//< ray entry
	float * ray_exit_d,				//< ray exit
	float * noise_d,				//< noise for ray offsets 
	float   samples_per_voxel		//< samples per voxel
	)
{
	dim3 block_size;
	block_size.x=16;
	block_size.y=16;
	dim3 grid_size;
	grid_size.x = iDivUp(n_u,block_size.x);
	grid_size.y = iDivUp(n_v,block_size.y);

	if (n_c==1)
		kernel_raycast<1><<<grid_size, block_size>>>(n_u, n_v, pixel_data_d, model_C_Pinv_d, voxel_data, ray_entry_d, ray_exit_d, noise_d, samples_per_voxel);
	else if (n_c==3)
		kernel_raycast<3><<<grid_size, block_size>>>(n_u, n_v, pixel_data_d, model_C_Pinv_d, voxel_data, ray_entry_d, ray_exit_d, noise_d, samples_per_voxel);
	else if (n_c==4)
		kernel_raycast<4><<<grid_size, block_size>>>(n_u, n_v, pixel_data_d, model_C_Pinv_d, voxel_data, ray_entry_d, ray_exit_d, noise_d, samples_per_voxel);
	else std::cerr << __FILE__ << " : " << __LINE__ << " : invalid number of channels!" << std::endl;

	hipDeviceSynchronize();
	cudaCheckState
}