#include "hip/hip_runtime.h"
// Created by A. Aichert on Fr Nov 29th 2013. Rewrite Tue Dec 19th 2017.
#include <iostream>

#include <LibUtilsCuda/CudaMemory.h>

__constant__ float Pi=3.14159265359;

// Make sure a point is within rectangle [0,0]-[n_u n_v]
__device__ inline bool inBounds(float u, float v, float n_u, float n_v)
{
	if (u<=n_u&&v<=n_v&&u>=0&&v>=0) return true;
	return false;
}

// A basic sorting algorithm of four values.
__device__ __host__ inline void sort4(float *v)
{
	for (int j=0;j<3;j++)
		for (int i=0;i<3;i++)
			if (v[i]>v[i+1])
			{
				float tmp=v[i];
				v[i]=v[i+1];
				v[i+1]=tmp;
			}
}

// Transform image into Radon space and coppute angular derivative
template <bool derivative>
__global__ void radonDerivative(hipTextureObject_t tex, float n_u, float n_v, float *out, int n_alpha, int n_t)
{
	// Compute bin index in image of Radon transform and corresponding line
	int   idx;
	float o[2];
	float d[2];
	{
		float l[3];
		int ix = blockIdx.x * blockDim.x + threadIdx.x;
		int iy = blockIdx.y * blockDim.y + threadIdx.y;
		if (ix>=n_alpha) return;
		if (iy>=n_t) return;
		idx=iy*n_alpha+ix;
		// Find index and relative location in range [-0.5, 0.5]^2
		float x_rel=(ix/(float)n_alpha-0.5f);
		float y_rel=(iy/(float)n_t    -0.5f);
		// Compute image diagonal d
		float diag=sqrtf(n_u*n_u+n_v*n_v);
		// Find line parameters
		float alpha=x_rel*Pi;    // angle alpha between -Pi/2 and Pi/2
		float tau  =y_rel*diag;  // distance t  between  -d/2 and  d/2
		// The line in Hessian normal form
		l[0]=-sinf(alpha);
		l[1]= cosf(alpha);
		l[2]=-tau;
		// Move coordinate origin to image center
		l[2]+=-0.5f*n_u*l[0]-0.5f*n_v*l[1];
		// Establish 1D line coordinate system (line coordinate t)
		// Line Origin : Closest point to the origin (t=0)
		o[0]=-l[2]*l[0];
		o[1]=-l[2]*l[1];
		// Line direction
		d[0]=l[1];
		d[1]=-l[0];
	}

	// Compute range at which the line intersects the image.
	float t,t_max;
	{
		float ts[]={
			(    1.f-o[0])/d[0],
			(n_u-1.f-o[0])/d[0],
			(    1.f-o[1])/d[1],
			(n_v-1.f-o[1])/d[1]
		};
		// Avoid Inf/NaN in case of vertical and horizontal lines.
		if (d[0]*d[0]<1e-12f) ts[0]=-(ts[1]=1e10f);
		if (d[1]*d[1]<1e-12f) ts[2]=-(ts[3]=1e10f);
		// Sorting - the middle two correspond to image edges.
		sort4(ts);
		t=ts[1];
		t_max=ts[2];		
	}

	__syncthreads();

	// Early exit if point for t_min (and hence no other) is within bounds
	if (!inBounds(o[0]+t*d[0],o[1]+t*d[1],n_u,n_v) || t_max <= t) {
		out[idx]=0;
		return;
	}

	//out[idx]=(t_max-t);
	//return;

	// Shift line origin since pixel[0][0] is at tex2D(.5,.5)
	o[0]+=.5f;
	o[1]+=.5f;

	// Compute line integral (or derivative thereof)
	const float step=.66;
	float sum=0;
	if (!derivative)
	{
		// Start summation over a line
		for (; t<=t_max; t+=step)
			sum +=tex2D<float>(tex,o[0]+t*d[0]     ,o[1]+t*d[1]     );
		out[idx]=sum*step;
	}
	else
	{
		// Move by half pixel in normal direction
		o[0]-=.5f*d[1];
		o[1]+=.5f*d[0];
		// Start summation over two parallel lines
		float sumo=0;
		for (; t<=t_max; t+=step)
		{
			sum +=tex2D<float>(tex,o[0]+t*d[0]     ,o[1]+t*d[1]     );
			// Offset by one pixel in negative normal direction
			sumo+=tex2D<float>(tex,o[0]+t*d[0]+d[1],o[1]+t*d[1]-d[0]);
		}
		// and return difference (approximation to derivative)
		out[idx]=(sum-sumo)*step;
	}

}

/// Computed Radon derivative and returns n_t*n_alpha array in out_d (device array).
void computeDerivLineIntegrals(hipTextureObject_t in, int n_u, int n_v, int n_alpha, int n_t, bool is_derivative, float *out_d)
{
	cudaCheckState
	// Threads per block and problem size
	dim3 block_size;
	block_size.x=32;
	block_size.y=4;
	dim3 grid_size;
	grid_size.x = iDivUp(n_alpha,block_size.x);
	grid_size.y = iDivUp(n_t,block_size.y);
	// Start kernel (radon transform)
	if (is_derivative)
		radonDerivative<true><<<grid_size, block_size>>>(in, (float)n_u, (float)n_v, out_d, n_alpha, n_t);
	else
		radonDerivative<false><<<grid_size, block_size>>>(in, (float)n_u, (float)n_v, out_d, n_alpha, n_t);
	hipDeviceSynchronize();
	cudaCheckState
}
