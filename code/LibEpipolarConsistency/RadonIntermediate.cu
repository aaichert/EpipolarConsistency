#include "hip/hip_runtime.h"
// Created by A. Aichert on Fr Nov 29th 2013. Rewrite Tue Dec 19th 2017.
#include <iostream>

#include <LibUtilsCuda/CudaMemory.h>

#include <hipfft/hipfft.h>

__constant__ float Pi=3.14159265359;

// Make sure a point is within rectangle [0,0]-[n_u n_v]
__device__ inline bool inBounds(float u, float v, float n_u, float n_v)
{
	if (u<=n_u&&v<=n_v&&u>=0&&v>=0) return true;
	return false;
}

// A basic sorting algorithm of four values.
__device__ __host__ inline void sort4(float *v)
{
	for (int j=0;j<3;j++)
		for (int i=0;i<3;i++)
			if (v[i]>v[i+1])
			{
				float tmp=v[i];
				v[i]=v[i+1];
				v[i+1]=tmp;
			}
}

// Transform image into Radon space and coppute angular derivative
template <bool derivative>
__global__ void radonDerivative(hipTextureObject_t tex, float n_u, float n_v, float *out, int n_alpha, int n_t, short post_process)
{
	// Compute bin index in image of Radon transform and corresponding line
	int   idx;
	float o[2];
	float d[2];
	{
		float l[3];
		int ix = blockIdx.x * blockDim.x + threadIdx.x;
		int iy = blockIdx.y * blockDim.y + threadIdx.y;
		if (ix>=n_alpha) return;
		if (iy>=n_t) return;
		idx=iy*n_alpha+ix;
		// Find index and relative location in range [-0.5, 0.5]^2
		float x_rel=(ix/(float)n_alpha-0.5f);
		float y_rel=(iy/(float)n_t    -0.5f);
		// Compute image diagonal d
		float diag=sqrtf(n_u*n_u+n_v*n_v);
		// Find line parameters
		float alpha=x_rel*Pi;    // angle alpha between -Pi/2 and Pi/2
		float tau  =y_rel*diag;  // distance t  between  -d/2 and  d/2
		// The line in Hessian normal form
		l[0]=-sinf(alpha);
		l[1]= cosf(alpha);
		l[2]=-tau;
		// Move coordinate origin to image center
		l[2]+=-0.5f*n_u*l[0]-0.5f*n_v*l[1];
		// Establish 1D line coordinate system (line coordinate t)
		// Line Origin : Closest point to the origin (t=0)
		o[0]=-l[2]*l[0];
		o[1]=-l[2]*l[1];
		// Line direction
		d[0]=l[1];
		d[1]=-l[0];
	}

	// Compute range over which the line intersects the image.
	float t,t_max;
	{
		float ts[]={
			(    1.f-o[0])/d[0],
			(n_u-1.f-o[0])/d[0],
			(    1.f-o[1])/d[1],
			(n_v-1.f-o[1])/d[1]
		};
		// Avoid Inf/NaN in case of vertical and horizontal lines.
		if (d[0]*d[0]<1e-12f) ts[0]=-(ts[1]=1e10f);
		if (d[1]*d[1]<1e-12f) ts[2]=-(ts[3]=1e10f);
		// Sorting - the middle two correspond to image edges.
		sort4(ts);
		t=ts[1];
		t_max=ts[2];		
	}

	__syncthreads();

	// Early exit if point for t_min (and hence no other) is within bounds
	if (!inBounds(o[0]+t*d[0],o[1]+t*d[1],n_u,n_v) || t_max <= t) {
		out[idx]=0;
		return;
	}

	//out[idx]=(t_max-t);
	//return;

	// Shift line origin since pixel[0][0] is at tex2D(.5,.5)
	o[0]+=.5f;
	o[1]+=.5f;

	// Compute line integral (or derivative thereof)
	const float step=.66;
	float sum=0;
	if (!derivative)
	{
		// Start summation over a line
		for (; t<=t_max; t+=step)
			sum +=tex2D<float>(tex,o[0]+t*d[0]     ,o[1]+t*d[1]     );
		out[idx]=sum*step;
	}
	else
	{
		// Move by half pixel in normal direction
		o[0]-=.5f*d[1];
		o[1]+=.5f*d[0];
		// Start summation over two parallel lines
		float sumo=0;
		for (; t<=t_max; t+=step)
		{
			sum +=tex2D<float>(tex,o[0]+t*d[0]     ,o[1]+t*d[1]     );
			// Offset by one pixel in negative normal direction
			sumo+=tex2D<float>(tex,o[0]+t*d[0]+d[1],o[1]+t*d[1]-d[0]);
		}
		// and return difference (approximation to derivative)
		if (post_process==1) {
			float result=(sum-sumo)*step;
			if (result<0)
				out[idx]=-sqrt(-result);
			else
				out[idx]=sqrt(result);
		}
		else if (post_process==2) {
			float result=(sum-sumo)*step;
			if (result<0)
				out[idx]=-log(-result+1);
			else
				out[idx]=log(result+1);
		}
		else
			out[idx]=(sum-sumo)*step;
	}

}

// Predcl of ramp filter for Smith DCC
void apply1DRampFilter(float* radon_transform_d, int n_alpha, int n_t);

/// Computed Radon intermediate function and returns n_t*n_alpha array in out_d (device array). Applied filter (Derivative=0, Ramp=1, None=2)
void computeDerivLineIntegrals(hipTextureObject_t in, int n_u, int n_v, int n_alpha, int n_t, int filter, int post_process, float *out_d)
{
	cudaCheckState
	// Threads per block and problem size
	dim3 block_size;
	block_size.x=32;
	block_size.y=4;
	dim3 grid_size;
	grid_size.x = iDivUp(n_alpha,block_size.x);
	grid_size.y = iDivUp(n_t,block_size.y);
	// Start kernel (radon transform)
	if (filter==0)
		radonDerivative<true><<<grid_size, block_size>>>(in, (float)n_u, (float)n_v, out_d, n_alpha, n_t, (short)post_process);
	else
		radonDerivative<false><<<grid_size, block_size>>>(in, (float)n_u, (float)n_v, out_d, n_alpha, n_t, (short)post_process);
	hipDeviceSynchronize();
	cudaCheckState

	if (filter==1)
		apply1DRampFilter(out_d, n_alpha, n_t);

}

// Multiply ramp and scale
__global__ void ramp_filter1D(hipfftComplex *ftrt, int n_alpha, int n_theta, float scale)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	if (ix>=n_alpha) return;
	if (iy>=n_theta) return;
	int idx=iy*n_alpha+ix;

	ftrt[idx].x*=iy*scale;
	ftrt[idx].y*=iy*scale;
}

///
void apply1DRampFilter(float* radon_transform_d, int n_alpha, int n_t)
{
	cudaCheckState

    // Device side output data allocation
    hipfftComplex *fourier_transforms_of_radon_transform_columns_d;
	int n_theta=n_t/2 + 1;
	hipMalloc((void**)&fourier_transforms_of_radon_transform_columns_d, n_theta*n_alpha*sizeof(hipfftComplex));
	cudaCheckState

	// n_alpha batched 1D FFTs over n_t elemenets in the SLOW image direction (hence a stride of n_alpha);
	hipfftHandle plan;
	hipfftPlanMany(&plan,
		1, &n_t,              // request 1D FFTs of size n_t
		&n_t, n_alpha, 1,     // stride and distance between batches (input) ; in this case interleaved storage, so stride is larger than batch distance.
		&n_theta, n_alpha, 1, // stride and distance between batches (output); in this case interleaved storage, so stride is larger than batch distance.
		HIPFFT_R2C,            // we convert real to complex
		n_alpha);             // number of FFTs
	cudaCheckState
    hipfftExecR2C(plan, radon_transform_d, fourier_transforms_of_radon_transform_columns_d);
	hipDeviceSynchronize();
    hipfftDestroy(plan);
	cudaCheckState

	// Threads per block and problem size
	dim3 block_size;
	block_size.x=8;
	block_size.y=32;
	dim3 grid_size;
	grid_size.x = iDivUp(n_alpha,block_size.x);
	grid_size.y = iDivUp(n_theta,block_size.y);

	// Kernel for multiplication with ramp and appropriate scaling.
	ramp_filter1D<<<grid_size, block_size>>>(fourier_transforms_of_radon_transform_columns_d, n_alpha, n_theta, -0.5f/(n_t*n_theta) );
	cudaCheckState

	// Execute inverse FFTs
	hipfftPlanMany(&plan,
		1, &n_t,              // request 1D FFTs of size n_t
		&n_theta, n_alpha, 1, // stride and distance between batches (input) ; in this case interleaved storage, so stride is larger than batch distance.
		&n_t, n_alpha, 1,     // stride and distance between batches (output); in this case interleaved storage, so stride is larger than batch distance.
		HIPFFT_C2R,            // we convert real to complex
		n_alpha);             // number of FFTs
	cudaCheckState

	// Clean up
	hipfftExecC2R(plan, fourier_transforms_of_radon_transform_columns_d, radon_transform_d);
	hipDeviceSynchronize();
    hipfftDestroy(plan);
    hipFree(fourier_transforms_of_radon_transform_columns_d);
	cudaCheckState
}

